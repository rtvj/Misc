#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#ifndef _FILTER_KERNEL_H_
#define _FILTER_KERNEL_H_


__global__ void SobelFilter(unsigned char* g_DataIn, unsigned char* g_DataOut, int width, int height)
{
   __shared__ unsigned char sharedMem[BLOCK_HEIGHT * BLOCK_WIDTH];
   float s_SobelMatrix[9];

    s_SobelMatrix[0] = -1;
    s_SobelMatrix[1] = 0;
    s_SobelMatrix[2] = 1;

    s_SobelMatrix[3] = -2;
    s_SobelMatrix[4] = 0;
    s_SobelMatrix[5] = 2;

    s_SobelMatrix[6] = -1;
    s_SobelMatrix[7] = 0;
    s_SobelMatrix[8] = 1;

   // Computer the X and Y global coordinates
   int x = blockIdx.x * TILE_WIDTH + threadIdx.x ;//- FILTER_RADIUS;
   int y = blockIdx.y * TILE_HEIGHT + threadIdx.y ;//- FILTER_RADIUS;

   // Get the Global index into the original image
   int index = y * (width) + x;

   // STUDENT:  Check 1
   // Handle the extra thread case where the image width or height 
   // 
   if (x >= width || y >= height)
      return;

   // STUDENT: Check 2
   // Handle the border cases of the global image
   if( x < FILTER_RADIUS || y < FILTER_RADIUS) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

   if ((x > width - FILTER_RADIUS - 1)&&(x <width)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

    if ((y > height - FILTER_RADIUS - 1)&&(y < height)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

   // Perform the first load of values into shared memory
   int sharedIndex = threadIdx.y * blockDim.y + threadIdx.x;
   sharedMem[sharedIndex] = g_DataIn[index];
   __syncthreads();


   // STUDENT: Make sure only the thread ids should write the sum of the neighbors.
                 float sumX = 0, sumY=0;
	if (threadIdx.x < FILTER_RADIUS || threadIdx.x >= (blockDim.x-FILTER_RADIUS))
     return;

   if (threadIdx.y < FILTER_RADIUS || threadIdx.y >= (blockDim.y-FILTER_RADIUS))
     return;       
				 
				 for(int dy = -FILTER_RADIUS; dy <= FILTER_RADIUS; dy++) {
					for(int dx = -FILTER_RADIUS; dx <= FILTER_RADIUS; dx++) {
					// float Pixel = (float)(sharedMem[y*width + x +  (dy * width + dx)]);
					float Pixel = (float)(sharedMem[(dy + threadIdx.y) * blockDim.y + (threadIdx.x + dx)]);
					 sumX += Pixel * s_SobelMatrix[(dx + FILTER_RADIUS) * FILTER_DIAMETER + (dy+FILTER_RADIUS)];
					 sumY += Pixel * s_SobelMatrix[(dy + FILTER_RADIUS) * FILTER_DIAMETER + (dx+FILTER_RADIUS)];
			 
			  //g_DataOut[index] = Pixel;
          }
        }
                 g_DataOut[index] = abs(sumX) + abs(sumY) > EDGE_VALUE_THRESHOLD ? 255 : 0;
}
__global__ void SobelFilter5(unsigned char* g_DataIn, unsigned char* g_DataOut, int width, int height, float* s_SobelMatrix)
{	
	
   __shared__ unsigned char sharedMem[BLOCK_HEIGHT * BLOCK_WIDTH];
  
   // Computer the X and Y global coordinates
   int x = blockIdx.x * TILE_WIDTH + threadIdx.x ;//- FILTER_RADIUS;
   int y = blockIdx.y * TILE_HEIGHT + threadIdx.y ;//- FILTER_RADIUS;

   // Get the Global index into the original image
   int index = y * (width) + x;

   // STUDENT:  Check 1
   // Handle the extra thread case where the image width or height 
   // 
   if (x >= width || y >= height)
      return;

   // STUDENT: Check 2
   // Handle the border cases of the global image
   if( x < FILTER_RADIUS || y < FILTER_RADIUS) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

   if ((x > width - FILTER_RADIUS - 1)&&(x <width)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

    if ((y > height - FILTER_RADIUS - 1)&&(y < height)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

   // Perform the first load of values into shared memory
   int sharedIndex = threadIdx.y * blockDim.y + threadIdx.x;
   sharedMem[sharedIndex] = g_DataIn[index];
   __syncthreads();


   // STUDENT: Make sure only the thread ids should write the sum of the neighbors.
if (threadIdx.x < FILTER_RADIUS || threadIdx.x >= (blockDim.x - FILTER_RADIUS)) 
     return;

   if (threadIdx.y < FILTER_RADIUS || threadIdx.y >= (blockDim.y - FILTER_RADIUS))
     return;
//taking in account only the blue region

   float sumX = 0, sumY=0;

       // sum up the 9 values to calculate both the x and y direction
       
       for(int dy = -FILTER_RADIUS; dy <= FILTER_RADIUS; dy++) 
	{
          for(int dx = -FILTER_RADIUS; dx <= FILTER_RADIUS; dx++) 
		{
           	  float Pixel = (float)(sharedMem[ (threadIdx.y * blockDim.y) + threadIdx.x + (dx * blockDim.y + dy)]);
           	  sumX += Pixel * s_SobelMatrix[(dy + FILTER_RADIUS) * FILTER_DIAMETER + (dx+FILTER_RADIUS)];
           	  sumY += Pixel * s_SobelMatrix[(dx + FILTER_RADIUS) * FILTER_DIAMETER + (dy+FILTER_RADIUS)];
        }
    }
g_DataOut[index] = abs(sumX) + abs(sumY) > EDGE_VALUE_THRESHOLD ? 255 : 0;
}

__global__ void AverageFilter(unsigned char* g_DataIn, unsigned char* g_DataOut, int width, int height)
{
    __shared__ unsigned char sharedMem[BLOCK_HEIGHT*BLOCK_WIDTH];

   int x = blockIdx.x * TILE_WIDTH + threadIdx.x ;//- FILTER_RADIUS;
   int y = blockIdx.y * TILE_HEIGHT + threadIdx.y ;//- FILTER_RADIUS;

   // Get the Global index into the original image
   int index = y * (width) + x;

  // STUDENT: write code for Average Filter : use Sobel as base code
float SUM = 0;
	
	if (x >= width || y >= height)
      return;
	  
	  
	 if( x < FILTER_RADIUS || y < FILTER_RADIUS) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

   if ((x > width - FILTER_RADIUS - 1)&&(x <width)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

    if ((y > height - FILTER_RADIUS - 1)&&(y < height)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

   // Perform the first load of values into shared memory
   int sharedIndex = threadIdx.y * blockDim.y + threadIdx.x;
   sharedMem[sharedIndex] = g_DataIn[index];
   __syncthreads();
   
      if (threadIdx.x < FILTER_RADIUS || threadIdx.x >= (blockDim.x-FILTER_RADIUS)) 
     return;

   if (threadIdx.y < FILTER_RADIUS || threadIdx.y >= (blockDim.y-FILTER_RADIUS))
     return;
   
   
  // STUDENT: write code for Average Filter : use Sobel as base code
 
   for(int dy = -FILTER_RADIUS; dy <= FILTER_RADIUS; dy++) 
    {
      for(int dx = -FILTER_RADIUS; dx <= FILTER_RADIUS; dx++) 
      {
         float Pixel = (float)(sharedMem[(dy + threadIdx.y) * blockDim.y + (threadIdx.x + dx)]);
		 SUM = SUM + Pixel; 
		 
      }
    }
					 SUM = SUM/FILTER_AREA;
					 g_DataOut[index] =  SUM;

}



__global__ void HighBoostFilter(unsigned char* g_DataIn, unsigned char* g_DataOut, int width, int height)
{
  __shared__ unsigned char sharedMem[BLOCK_HEIGHT*BLOCK_WIDTH];

  int x = blockIdx.x * TILE_WIDTH + threadIdx.x ;//- FILTER_RADIUS;
  int y = blockIdx.y * TILE_HEIGHT + threadIdx.y ;//- FILTER_RADIUS;

  // Get the Global index into the original image
  int index = y * (width) + x;


  // STUDENT: write code for High Boost Filter : use Sobel as base code
  int SUM = 0;

        if (x >= width || y >= height)
      return;


         if( x < FILTER_RADIUS || y < FILTER_RADIUS) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

   if ((x > width - FILTER_RADIUS - 1)&&(x <width)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

    if ((y > height - FILTER_RADIUS - 1)&&(y < height)) {
       g_DataOut[index] = g_DataIn[index];
       return;
    }

   // Perform the first load of values into shared memory
   int sharedIndex = threadIdx.y * blockDim.y + threadIdx.x;
   sharedMem[sharedIndex] = g_DataIn[index];
   __syncthreads();

	float px = 0.0;
   if (threadIdx.x < FILTER_RADIUS || threadIdx.x >= (blockDim.x-FILTER_RADIUS)) 
     return;

   if (threadIdx.y < FILTER_RADIUS || threadIdx.y >= (blockDim.y-FILTER_RADIUS))
     return;

  // STUDENT: write code for High Boost Filter : use Sobel as base code
  				for(int dx = -FILTER_RADIUS; dx <= FILTER_RADIUS; dx++)
				for(int dy = -FILTER_RADIUS; dy <= FILTER_RADIUS; dy++) 
				{
					{
					// float Pixel = (float)(g_DataIn[y*width + x +  (dy * width + dx)]);
					float Pixel = (float)(sharedMem[(dy + threadIdx.y) * blockDim.y + (threadIdx.x + dx)]);
					 SUM = SUM + Pixel;
					// 
					}
				}
					 SUM = SUM/9;
					 
					  px = sharedMem[(threadIdx.y) * blockDim.y + (threadIdx.x)];
					// g_DataOut[index] = SUM;
					g_DataOut[index] = CLAMP_8bit((int)(px + HIGH_BOOST_FACTOR*(unsigned char)(px-SUM)));
					//g_DataOut[index] = SUM;
}


#endif // _FILTER_KERNEL_H_


